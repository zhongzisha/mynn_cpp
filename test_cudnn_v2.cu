#include "hip/hip_runtime.h"

#include <glog/logging.h>
#include <pthread.h>

#include <sstream>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <vector>
using namespace std;

#include "boost/lexical_cast.hpp"
#include "boost/algorithm/string.hpp"
#include "google/protobuf/text_format.h"
#include <boost/filesystem.hpp>
using namespace boost;

#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand.h>

extern "C" {
#include <cblas.h>
}


#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/highgui/highgui_c.h>
#include <opencv2/imgproc/imgproc.hpp>
using namespace cv;

#include "matio.h"

#include "myproto.pb.h"
#include "io.hpp"
#include "db.hpp"
#include "internal_thread.hpp"


template <typename Dtype> enum matio_types matio_type_map();
template <> enum matio_types matio_type_map<float>() { return MAT_T_SINGLE; }
template <> enum matio_types matio_type_map<double>() { return MAT_T_DOUBLE; }
template <> enum matio_types matio_type_map<int>() { return MAT_T_INT32; }
template <> enum matio_types matio_type_map<unsigned int>() { return MAT_T_UINT32; }

template <typename Dtype> enum matio_classes matio_class_map();
template <> enum matio_classes matio_class_map<float>() { return MAT_C_SINGLE; }
template <> enum matio_classes matio_class_map<double>() { return MAT_C_DOUBLE; }
template <> enum matio_classes matio_class_map<int>() { return MAT_C_INT32; }
template <> enum matio_classes matio_class_map<unsigned int>() { return MAT_C_UINT32; }

const char* curandGetErrorString(hiprandStatus_t error) {
	switch (error) {
	case HIPRAND_STATUS_SUCCESS:
		return "HIPRAND_STATUS_SUCCESS";
	case HIPRAND_STATUS_VERSION_MISMATCH:
		return "HIPRAND_STATUS_VERSION_MISMATCH";
	case HIPRAND_STATUS_NOT_INITIALIZED:
		return "HIPRAND_STATUS_NOT_INITIALIZED";
	case HIPRAND_STATUS_ALLOCATION_FAILED:
		return "HIPRAND_STATUS_ALLOCATION_FAILED";
	case HIPRAND_STATUS_TYPE_ERROR:
		return "HIPRAND_STATUS_TYPE_ERROR";
	case HIPRAND_STATUS_OUT_OF_RANGE:
		return "HIPRAND_STATUS_OUT_OF_RANGE";
	case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
		return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
	case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
		return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
	case HIPRAND_STATUS_LAUNCH_FAILURE:
		return "HIPRAND_STATUS_LAUNCH_FAILURE";
	case HIPRAND_STATUS_PREEXISTING_FAILURE:
		return "HIPRAND_STATUS_PREEXISTING_FAILURE";
	case HIPRAND_STATUS_INITIALIZATION_FAILED:
		return "HIPRAND_STATUS_INITIALIZATION_FAILED";
	case HIPRAND_STATUS_ARCH_MISMATCH:
		return "HIPRAND_STATUS_ARCH_MISMATCH";
	case HIPRAND_STATUS_INTERNAL_ERROR:
		return "HIPRAND_STATUS_INTERNAL_ERROR";
	}
	return "Unknown hiprand status";
}

const char* cublasGetErrorString(hipblasStatus_t error) {
	switch (error) {
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";
	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";
	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";
	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";
	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";
	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";
	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";
	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";
#if CUDA_VERSION >= 6000
	case HIPBLAS_STATUS_NOT_SUPPORTED:
		return "HIPBLAS_STATUS_NOT_SUPPORTED";
#endif
#if CUDA_VERSION >= 6050
	case HIPBLAS_STATUS_UNKNOWN:
		return "HIPBLAS_STATUS_UNKNOWN";
#endif
	}
	return "Unknown cublas status";
}

/********************************************************
 * Prints the error message, and exits
 * ******************************************************/


// CUDA: various checks for different function calls.
#define CUDA_CHECK(condition) \
		/* Code block avoids redefinition of hipError_t error */ \
		do { \
			hipError_t error = condition; \
			if (error != hipSuccess) \
			printf("CUDA Error: %s\n", hipGetErrorString(error)); \
		} while (0)

#define CUBLAS_CHECK(condition) \
		do { \
			hipblasStatus_t status = condition; \
			if (status != HIPBLAS_STATUS_SUCCESS) \
			printf("CUBLAS Error: %s\n", cublasGetErrorString(status)); \
		} while (0)

#define CURAND_CHECK(condition) \
		do { \
			hiprandStatus_t status = condition; \
			if (status != HIPRAND_STATUS_SUCCESS) \
			printf("CURAND Error: %s\n", curandGetErrorString(status)); \
		} while (0)

#define CUDNN_CHECK(status) \
		do { \
			if (status != HIPDNN_STATUS_SUCCESS) \
			printf("CUDNN Error: %s\n", hipdnnGetErrorString(status)); \
		} while (0)

#define EXIT_WAIVED 0
#define FatalError(s) {                                                \
		std::stringstream _where, _message;                                \
		_where << __FILE__ << ':' << __LINE__;                             \
		_message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;\
		std::cerr << _message.str() << "\nAborting...\n";                  \
		hipDeviceReset();                                                 \
		exit(EXIT_FAILURE);                                                \
		}


void EnableP2P(vector<int> gpus)
{
	// check p2p access
	hipDeviceProp_t prop[gpus.size()];
	for(int i = 0; i < gpus.size(); i++) {
		hipGetDeviceProperties(&prop[i], gpus[i]);
	}

	for(int i = 0; i < gpus.size(); i++) {
		for(int j = 0; j < gpus.size(); j++) {
			if(i==j)
				continue;
			int can_access_peer;
			hipDeviceCanAccessPeer(&can_access_peer, gpus[i], gpus[j]);
			if(can_access_peer) {
				hipSetDevice(gpus[i]);
				hipDeviceEnablePeerAccess(gpus[j], 0);
				hipSetDevice(gpus[j]);
				hipDeviceEnablePeerAccess(gpus[i], 0);
				const bool has_uva = (prop[gpus[i]].unifiedAddressing && prop[gpus[j]].unifiedAddressing);
				if(has_uva) {
					printf("(%d <--> %d): YES!\n", gpus[i], gpus[j]);
				}
			} else {
				printf("(%d <--> %d): NO!\n", gpus[i], gpus[j]);
			}
		}
	}
}

void DisableP2P(vector<int> gpus)
{
	for(int i = 0; i < gpus.size(); i++) {
		hipSetDevice(gpus[i]);
		hipDeviceDisablePeerAccess(gpus[i]);
	}
}

// CUDA: thread number configuration.
// Use 1024 threads per block, which requires cuda sm_2x or above,
// or fall back to attempt compatibility (best of luck to you).
#if __CUDA_ARCH__ >= 200
    const int GPU_CUDA_NUM_THREADS = 1024;
#else
    const int GPU_CUDA_NUM_THREADS = 512;
#endif

// CUDA: number of blocks for threads.
inline int GPU_GET_BLOCKS(const int N) {
  return (N + GPU_CUDA_NUM_THREADS - 1) / GPU_CUDA_NUM_THREADS;
}

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

__global__ void add_kernel(const int n, const float* a,
    const float* b, float* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

void cpu_add(const int N, const float *a, const float *b, float *y) {
#pragma omp parallel
	for(int i = 0; i < N; i++) {
		y[i] = a[i] + b[i];
	}
}

void gpu_add(const int N, const float* a, const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<<<GPU_GET_BLOCKS(N), GPU_CUDA_NUM_THREADS>>>(N, a, b, y);
}

void gpu_copy(const int N, const float *X, float *Y) {
	CUDA_CHECK( hipMemcpy(Y, X, sizeof(float) * N, hipMemcpyDefault) );
}

void gpu_scal(hipblasHandle_t cublashandle, const int N, const float alpha, float *X) {
	CUBLAS_CHECK( hipblasSscal(cublashandle, N, &alpha, X, 1) );
}

void gpu_axpy(hipblasHandle_t cublashandle, const int N, const float alpha,
		const float* X, float* Y) {
	CUBLAS_CHECK( hipblasSaxpy(cublashandle, N, &alpha, X, 1, Y, 1) );
}

void gpu_axpby(hipblasHandle_t cublashandle, const int N, const float alpha,
		const float* X, const float beta, float* Y) {
	gpu_scal(cublashandle, N, beta, Y);
	gpu_axpy(cublashandle, N, alpha, X, Y);
}

void gpu_gemv(hipblasHandle_t cublashandle,
		const CBLAS_TRANSPOSE TransA, const int M,
		const int N, const float alpha, const float* A, const float* x,
		const float beta, float* y) {
	hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	CUBLAS_CHECK( hipblasSgemv(cublashandle, cuTransA, N, M, &alpha, A, N, x, 1, &beta, y, 1) );
}

void gpu_gemm(hipblasHandle_t cublashandle,
		const CBLAS_TRANSPOSE TransA,
		const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
		const float alpha, const float* A, const float* B, const float beta,
		float* C) {
	// Note that cublas follows fortran order.
	int lda = (TransA == CblasNoTrans) ? K : M;
	int ldb = (TransB == CblasNoTrans) ? N : K;
	hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
	hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
	CUBLAS_CHECK( hipblasSgemm(cublashandle, cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N) );
}

class Blob_t
{
public:
	int N;
	int C;
	int H;
	int W;
	float *data_cpu, *data_gpu;
	float *diff_cpu, *diff_gpu;

	Blob_t() : data_cpu(NULL), data_gpu(NULL), diff_cpu(NULL), diff_gpu(NULL), N(0), C(0), H(0), W(0) {};

	Blob_t(int N_, int C_, int H_, int W_) : data_cpu(NULL), data_gpu(NULL), diff_cpu(NULL), diff_gpu(NULL)
	{
		N = N_;
		C = C_;
		H = H_;
		W = W_;
	}

	~Blob_t()
	{
		if(data_cpu != NULL)
		{
			CUDA_CHECK( hipHostFree(data_cpu) );
			data_cpu = NULL;
		}
		if(data_gpu != NULL)
		{
			CUDA_CHECK( hipFree(data_gpu) );
			data_gpu = NULL;
		}
		if(diff_cpu != NULL)
		{
			CUDA_CHECK( hipHostFree(diff_cpu) );
			diff_cpu = NULL;
		}
		if(diff_gpu != NULL)
		{
			CUDA_CHECK( hipFree(diff_gpu) );
			diff_gpu = NULL;
		}
	}

	inline int count() const {
		return N * C * H * W;
	}
	inline int offset(const int n, const int c = 0, const int h = 0, const int w = 0) const {
		return ((n * C + c) * H + h) * W + w;
	}

	void print_gpu_data() {
		if(data_gpu == NULL)
			printf("gpu data is NULL.\n");
		data_to_cpu();
		for(int n = 0; n < N; n++) {
			for(int c = 0; c < C; c++) {
				for(int h = 0; h < H; h++) {
					for(int w = 0; w < W; w++) {
						int index = (((n)*C + c)*H + h)*W+w;
						printf("(%d, %d, %d, %d) : %.6f\n", n, c, h, w, data_cpu[index]);
					}
				}
			}
		}

	}

	void print_gpu_data(int howmany) {
		if(data_gpu == NULL)
			printf("gpu data is NULL.\n");
		data_to_cpu();
		for(int n = 0; n < 1; n++) {
			for(int c = 0; c < 1; c++) {
				for(int h = 0; h < 1; h++) {
					for(int w = 0; w < W; w++) {
						int index = (((n)*C + c)*H + h)*W+w;
						printf("(%d, %d, %d, %d) : %.6f\n", n, c, h, w, data_cpu[index]);
					}
				}
			}
		}
	}

	void print_cpu_data(int howmany) {
		if(data_cpu == NULL)
			printf("data_cpu is NULL.\n");
		for(int n = 0; n < 1; n++) {
			for(int c = 0; c < 1; c++) {
				for(int h = 0; h < 1; h++) {
					for(int w = 0; w < W; w++) {
						int index = (((n)*C + c)*H + h)*W+w;
						printf("(%d, %d, %d, %d) : %.6f\n", n, c, h, w, data_cpu[index]);
					}
				}
			}
		}
	}

	void save_cpu_data_and_diff_to_mat(const char *fname)
	{
		// save results into matlab format
		mat_t *matfp = Mat_Create(fname, 0);
		//matfp = Mat_CreateVer(fname, 0, MAT_FT_MAT73);
		size_t dims[4];
		dims[0] = W;
		dims[1] = H;
		dims[2] = C;
		dims[3] = N;
		matvar_t *matvar, *matvar2;
		// save data
		{
			matvar = Mat_VarCreate("data", matio_class_map<float>(), matio_type_map<float>(), 4, dims, data_cpu, 0);
			if(matvar == NULL)
				cout << "Error creating 'data' variable";
			matvar2 = Mat_VarCreate("diff", matio_class_map<float>(), matio_type_map<float>(), 4, dims, diff_cpu, 0);
			if(matvar2 == NULL)
				cout << "Error creating 'diff' variable";
			if(Mat_VarWrite(matfp, matvar, MAT_COMPRESSION_NONE) != 0)
				cout << "Error saving array 'data' into MAT file " << fname;

			if(Mat_VarWrite(matfp, matvar2, MAT_COMPRESSION_NONE) != 0)
				cout << "Error saving array 'diff' into MAT file " << fname;

			Mat_VarFree(matvar);
			Mat_VarFree(matvar2);
		}
		Mat_Close(matfp);
	}

	/*
	 *  data allocate
	 */
	void allocate_gpu_data()
	{
		int count = N * C * H * W;
		if(data_gpu != NULL)
			CUDA_CHECK( hipFree(data_gpu) );
		CUDA_CHECK( hipMalloc((void**)&data_gpu, count * sizeof(float)) );
		CUDA_CHECK( hipMemset(data_gpu, 0, count * sizeof(float)) );
	}

	void allocate_gpu_diff()
	{
		int count = N * C * H * W;
		if(diff_gpu != NULL)
			CUDA_CHECK( hipFree(diff_gpu) );
		CUDA_CHECK( hipMalloc((void**)&diff_gpu, count * sizeof(float)) );
		CUDA_CHECK( hipMemset(diff_gpu, 0, count * sizeof(float)) );
	}

	void allocate_cpu_data()
	{
		int count = N * C * H * W;
		if(data_cpu != NULL)
			CUDA_CHECK( hipHostFree(data_cpu) );
		CUDA_CHECK( hipHostMalloc((void**)&data_cpu, count * sizeof(float)) );
		CUDA_CHECK( hipMemset(data_cpu, 0, count * sizeof(float)) );
	}

	void allocate_cpu_diff()
	{
		int count = N * C * H * W;
		if(diff_cpu != NULL)
			CUDA_CHECK( hipHostFree(diff_cpu) );
		CUDA_CHECK( hipHostMalloc((void**)&diff_cpu, count * sizeof(float)) );
		CUDA_CHECK( hipMemset(diff_cpu, 0, count * sizeof(float)) );
	}

	/*
	 * data copy
	 */
	void data_to_gpu()
	{
		int count = N * C * H * W;
		if(data_gpu == NULL)
			CUDA_CHECK( hipMalloc((void**)&data_gpu, count * sizeof(float)) );
		if(data_cpu != NULL)
			CUDA_CHECK( hipMemcpy(data_gpu, data_cpu, count * sizeof(float), hipMemcpyHostToDevice) );
	}

	void diff_to_gpu()
	{
		int count = N * C * H * W;
		if(diff_gpu == NULL)
			CUDA_CHECK( hipMalloc((void**)&diff_gpu, count * sizeof(float)) );
		if(diff_cpu != NULL)
			CUDA_CHECK( hipMemcpy(diff_gpu, diff_cpu, count * sizeof(float), hipMemcpyHostToDevice) );
	}

	void data_to_cpu()
	{
		int count = N * C * H * W;
		if(data_cpu == NULL)
			CUDA_CHECK( hipHostMalloc((void**)&data_cpu, count * sizeof(float)) );
		if(data_gpu != NULL)
			CUDA_CHECK( hipMemcpy(data_cpu, data_gpu, count * sizeof(float), hipMemcpyDeviceToHost) );
	}

	void diff_to_cpu()
	{
		int count = N * C * H * W;
		if(diff_cpu == NULL)
			CUDA_CHECK( hipHostMalloc((void**)&diff_cpu, count * sizeof(float)) );
		if(diff_gpu != NULL)
			CUDA_CHECK( hipMemcpy(diff_cpu, diff_gpu, count * sizeof(float), hipMemcpyDeviceToHost) );
	}
};

void cpu_fill(float *ptr, int count, float value) {
	memset(ptr, value, count * sizeof(float));
}

void gpu_fill(hiprandGenerator_t curand_generator, float *ptr, int count, float mu, float std = 0.0f) {
	if(std == 0.0f) {
		CUDA_CHECK( hipMemset(ptr, mu, count * sizeof(float)) );
	} else {
		CURAND_CHECK( hiprandGenerateNormal(curand_generator, ptr, count, mu, std) );
	}
}

void CopyBlobData_gpu(const Blob_t *src, int src_gpu_id, Blob_t *dst, int dst_gpu_id)
{
	int count = src->count();
	if(src_gpu_id == dst_gpu_id) {
		hipSetDevice(src_gpu_id);
		hipMemcpy(dst->data_gpu, src->data_gpu, count * sizeof(float), hipMemcpyDefault);
	} else {
		hipDeviceProp_t prop[2];
		hipGetDeviceProperties(&prop[0], src_gpu_id);
		hipGetDeviceProperties(&prop[1], dst_gpu_id);
		int can_access_peer;
		hipDeviceCanAccessPeer(&can_access_peer, src_gpu_id, dst_gpu_id);
		if(can_access_peer) {
			const bool has_uva = (prop[0].unifiedAddressing && prop[1].unifiedAddressing);
			if(has_uva) {
				hipMemcpy(dst->data_gpu, src->data_gpu, count * sizeof(float), hipMemcpyDefault);
				return;
			}
		}

		float *temp_data = NULL;
		for(int i = 0; i <= 1; i++) {
			hipSetDevice(src_gpu_id);
			hipHostMalloc((void **)&temp_data, count * sizeof(float));
			hipMemcpy(temp_data, src->data_gpu, count * sizeof(float), hipMemcpyDeviceToHost);
			hipSetDevice(dst_gpu_id);
			hipMemcpy(dst->data_gpu, temp_data, count * sizeof(float), hipMemcpyHostToDevice);
			hipHostFree(temp_data);
		}
	}
}

void AddBlobDiff_gpu(const Blob_t *src, int src_gpu_id, Blob_t *dst, int dst_gpu_id)
{
	int count = src->count();
	if(src_gpu_id == dst_gpu_id) {
		hipSetDevice(src_gpu_id);
		gpu_add(count, src->diff_gpu, dst->diff_gpu, dst->diff_gpu);
	} else {
		hipDeviceProp_t prop[2];
		hipGetDeviceProperties(&prop[0], src_gpu_id);
		hipGetDeviceProperties(&prop[1], dst_gpu_id);
		int can_access_peer;
		hipDeviceCanAccessPeer(&can_access_peer, src_gpu_id, dst_gpu_id);
		if(can_access_peer) {
			const bool has_uva = (prop[0].unifiedAddressing && prop[1].unifiedAddressing);
			if(has_uva) {
				gpu_add(count, src->diff_gpu, dst->diff_gpu, dst->diff_gpu);
				return;
			}
		}

		float *temp_data = NULL;
		float *dst_temp_data = NULL;
		for(int i = 0; i <= 1; i++) {
			hipSetDevice(src_gpu_id);
			hipHostMalloc((void **)&temp_data, count * sizeof(float));
			hipMemcpy(temp_data, src->diff_gpu, count * sizeof(float), hipMemcpyDeviceToHost);
			hipSetDevice(dst_gpu_id);
			hipMalloc((void **)&dst_temp_data, count * sizeof(float));
			hipMemcpy(dst_temp_data, temp_data, count * sizeof(float), hipMemcpyHostToDevice);
			gpu_add(count, dst_temp_data, dst->diff_gpu, dst->diff_gpu);
			hipHostFree(temp_data);
			hipFree(dst_temp_data);
		}
	}
}

class DataLayerParameter_t
{
public:
	string backend;
	string source;
	string mean_file;
	int batch_size;
};

class DataLayer_t : public InternalThread
{
public:
	DataLayerParameter_t *data_params;
	Blob_t *prefetch_data_;
	Blob_t *prefetch_label_;
	Blob_t *mean_;
	int datum_size_;

	shared_ptr<db::DB> db_;
	shared_ptr<db::Cursor> cursor_;

	DataLayer_t(const DataLayerParameter_t *data_params_) {
		data_params = const_cast<DataLayerParameter_t *>(data_params_);

		prefetch_data_ = NULL;
		prefetch_label_ = NULL;
		mean_ = NULL;
		datum_size_ = 0;
	}

	~DataLayer_t() {
		JoinPrefetchThread(); // here, we should stop the final thread, when we delete the class instance
		delete prefetch_data_;
		delete prefetch_label_;
		delete mean_;
	}

	void Setup() {
		// Initialize DB
		db_.reset(db::GetDB(data_params->backend));
		db_->Open(data_params->source, db::READ);
		cursor_.reset(db_->NewCursor());

		// Read a data point, and use it to initialize the top blob.
		Datum datum;
		datum.ParseFromString(cursor_->value());
		datum_size_ = datum.channels() * datum.height() * datum.width();

		// printf("datum: (%d, %d, %d, %d)\n", data_params->batch_size, datum.channels(), datum.height(), datum.width());

		prefetch_data_ = new Blob_t(data_params->batch_size, datum.channels(), datum.height(), datum.width());
		prefetch_label_ = new Blob_t(data_params->batch_size, 1, 1, 1);
		prefetch_data_->allocate_cpu_data();
		prefetch_label_->allocate_cpu_data();

		mean_ = new Blob_t(1, datum.channels(), datum.height(), datum.width());
		mean_->allocate_cpu_data();
		BlobProto blob_proto;
		ReadProtoFromBinaryFileOrDie(data_params->mean_file.c_str(), &blob_proto);
		for (int i = 0; i < mean_->count(); ++i) {
			mean_->data_cpu[i] = (float)blob_proto.data(i);
		}
		// printf("mean_: (%d, %d, %d, %d)\n", mean_->N, mean_->C, mean_->H, mean_->W);

		CreatePrefetchThread();
	}

	void Forward_cpu(Blob_t *top_data, Blob_t *top_label) {
		// printf("First, join the thread.\n");
		JoinPrefetchThread();

		// printf("copy data to top_data.\n");
		top_data->N = prefetch_data_->N;
		top_data->C = prefetch_data_->C;
		top_data->H = prefetch_data_->H;
		top_data->W = prefetch_data_->W;
		top_data->allocate_cpu_data();
		memcpy(top_data->data_cpu, prefetch_data_->data_cpu, prefetch_data_->count() * sizeof(float));

		// printf("copy label to top_label.\n");
		top_label->N = prefetch_label_->N;
		top_label->C = prefetch_label_->C;
		top_label->H = prefetch_label_->H;
		top_label->W = prefetch_label_->W;
		top_label->allocate_cpu_data();
		memcpy(top_label->data_cpu, prefetch_label_->data_cpu, prefetch_label_->count() * sizeof(float));

		// printf("Start a new prefetch thread.\n");
		CreatePrefetchThread();
	}

	void Forward_cpu_multi(vector<Blob_t *> &top_data, vector<Blob_t *> &top_label, vector<int> &batch_sizes) {
		// printf("First, join the thread.\n");
		JoinPrefetchThread();

		for(int i = 0; i < batch_sizes.size(); i++) {
			int start_index = 0;
			for(int j = 0; j < i; j++) {
				start_index += batch_sizes[j];
			}
			// printf("copy data to top_data.\n");
			top_data[i]->N = batch_sizes[i];
			top_data[i]->C = prefetch_data_->C;
			top_data[i]->H = prefetch_data_->H;
			top_data[i]->W = prefetch_data_->W;
			top_data[i]->allocate_cpu_data();
			memcpy(top_data[i]->data_cpu,
					prefetch_data_->data_cpu + start_index * top_data[i]->C * top_data[i]->H * top_data[i]->W,
					top_data[i]->count() * sizeof(float));

			// printf("copy label to top_label.\n");
			top_label[i]->N = batch_sizes[i];
			top_label[i]->C = prefetch_label_->C;
			top_label[i]->H = prefetch_label_->H;
			top_label[i]->W = prefetch_label_->W;
			top_label[i]->allocate_cpu_data();
			memcpy(top_label[i]->data_cpu,
					prefetch_label_->data_cpu + start_index * top_data[i]->C * top_data[i]->H * top_data[i]->W,
					top_label[i]->count() * sizeof(float));
		}
		// printf("Start a new prefetch thread.\n");
		CreatePrefetchThread();
	}

protected:
	void CreatePrefetchThread() {
		CHECK(StartInternalThread()) << "Thread execution failed";
	}
	void JoinPrefetchThread() {
		CHECK(WaitForInternalThreadToExit()) << "Thread joining failed";
	}
	void InternalThreadEntry(){

		float *top_data = prefetch_data_->data_cpu;
		float *top_label = prefetch_label_->data_cpu;
		float *mean_data = mean_->data_cpu;
		for (int item_id = 0; item_id < data_params->batch_size; ++item_id) {

			// get a blob
			Datum datum;
			datum.ParseFromString(cursor_->value());

			// read one data
			const string& data = datum.data();
			if (data.size()) {
				for (int j = 0; j < datum_size_; ++j) {
					top_data[item_id * datum_size_ + j] = (static_cast<float>((uint8_t)data[j])) - mean_data[j];
				}
			} else {
				for (int j = 0; j < datum_size_; ++j) {
					top_data[item_id * datum_size_ + j] = (datum.float_data(j)) - mean_data[j];
				}
			}

			// read the label
			top_label[item_id] = datum.label();

			// go to the next iter
			cursor_->Next();
			if (!cursor_->valid()) {
				cursor_->SeekToFirst();
			}
		}
	}
};

class ConvolutionParameter_t
{
public:
	int filter_N;
	int filter_C;
	int filter_H;
	int filter_W;
	int pad_h, pad_w;
	int stride_h, stride_w;
	int upscale_h, upscale_w;
	hipdnnConvolutionMode_t cudnn_conv_mode;
};

class Layer_t
{
public:
	hipdnnDataType_t dataType;
	hipdnnTensorFormat_t tensorFormat;

	hipdnnHandle_t cudnnHandle;
	hipdnnTensorDescriptor_t bottomTensorDesc;
	hipdnnTensorDescriptor_t topTensorDesc;
	Layer_t()
	{
		dataType = HIPDNN_DATA_FLOAT;
		tensorFormat = HIPDNN_TENSOR_NCHW;
		cudnnHandle = NULL;

		CUDNN_CHECK( hipdnnCreate(&cudnnHandle) );
		CUDNN_CHECK( hipdnnCreateTensorDescriptor(&bottomTensorDesc) );
		CUDNN_CHECK( hipdnnCreateTensorDescriptor(&topTensorDesc) );
	}

	~Layer_t()
	{
		CUDNN_CHECK( hipdnnDestroyTensorDescriptor(bottomTensorDesc) );
		CUDNN_CHECK( hipdnnDestroyTensorDescriptor(topTensorDesc) );
		CUDNN_CHECK( hipdnnDestroy(cudnnHandle) );
	}

	void Setup(const Blob_t *bottom, Blob_t *top) {};
	void Forward(const Blob_t *bottom, Blob_t *top) {};
	void Backward(const Blob_t *top, Blob_t *bottom) {};
};

class ConvolutionLayer_t : public Layer_t
{
public:
	Blob_t *filtersBlob;
	Blob_t *biasBlob;

	hipdnnFilterDescriptor_t filterDesc;
	hipdnnTensorDescriptor_t biasTensorDesc;
	hipdnnConvolutionDescriptor_t convDesc;
	ConvolutionParameter_t *conv_params;


	ConvolutionLayer_t(const ConvolutionParameter_t *conv_params_)
	{
		conv_params = const_cast<ConvolutionParameter_t*>(conv_params_);
		filtersBlob = new Blob_t(conv_params->filter_N, conv_params->filter_C, conv_params->filter_H, conv_params->filter_W);
		biasBlob = new Blob_t(1, conv_params->filter_C, 1, 1);

		filtersBlob->allocate_gpu_data();
		filtersBlob->allocate_gpu_diff();
		biasBlob->allocate_gpu_data();
		biasBlob->allocate_gpu_diff();

		CUDNN_CHECK( hipdnnCreateFilterDescriptor(&filterDesc) );
		CUDNN_CHECK( hipdnnCreateTensorDescriptor(&biasTensorDesc) );
		CUDNN_CHECK( hipdnnCreateConvolutionDescriptor(&convDesc) );
	};

	~ConvolutionLayer_t()
	{
		delete filtersBlob; filtersBlob = NULL;
		delete biasBlob; biasBlob = NULL;

		CUDNN_CHECK( hipdnnDestroyConvolutionDescriptor(convDesc) );
		CUDNN_CHECK( hipdnnDestroyFilterDescriptor(filterDesc) );
		CUDNN_CHECK( hipdnnDestroyTensorDescriptor(biasTensorDesc) );
	}


	void Setup(const Blob_t *bottom, Blob_t *top)
	{

		CUDNN_CHECK( hipdnnSetTensor4dDescriptor(bottomTensorDesc,
				tensorFormat,
				dataType,
				bottom->N,
				bottom->C,
				bottom->H,
				bottom->W) );

		CUDNN_CHECK( hipdnnSetFilter4dDescriptor(filterDesc,
				dataType,
				filtersBlob->C,
				filtersBlob->N,
				filtersBlob->H,
				filtersBlob->W) );

		CUDNN_CHECK( hipdnnSetConvolution2dDescriptor(convDesc,
				conv_params->pad_h, // padding
				conv_params->pad_w,
				conv_params->stride_h, // stride
				conv_params->stride_w,
				conv_params->upscale_h, // upscale
				conv_params->upscale_w,
				conv_params->cudnn_conv_mode) );

		// find dimension of convolution output
		CUDNN_CHECK( hipdnnGetConvolution2dForwardOutputDim(convDesc,
				bottomTensorDesc,
				filterDesc,
				&(top->N),
				&(top->C),
				&(top->H),
				&(top->W)) );

		CUDNN_CHECK( hipdnnSetTensor4dDescriptor(topTensorDesc,
				tensorFormat,
				dataType,
				top->N,
				top->C,
				top->H,
				top->W) );

		top->allocate_gpu_data();
		top->allocate_gpu_diff();
	}

	void Forward(const Blob_t *bottom, Blob_t *top)
	{
		hipdnnConvolutionFwdAlgo_t algo;
		CUDNN_CHECK( hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
				bottomTensorDesc,
				filterDesc,
				convDesc,
				topTensorDesc,
				HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
				0,
				&algo ) );

		size_t sizeInBytes=0;
		void* workSpace=NULL;
		CUDNN_CHECK( hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
				bottomTensorDesc,
				filterDesc,
				convDesc,
				topTensorDesc,
				algo,
				&sizeInBytes) );
		if (sizeInBytes!=0)
		{
			CUDA_CHECK( hipMalloc(&workSpace,sizeInBytes) );
		}
		float alpha = float(1);
		float beta  = float(0);
		CUDNN_CHECK( hipdnnConvolutionForward(cudnnHandle,
				&alpha,
				bottomTensorDesc,
				bottom->data_gpu,
				filterDesc,
				filtersBlob->data_gpu,
				convDesc,
				algo,
				workSpace,
				sizeInBytes,
				&beta,
				topTensorDesc,
				top->data_gpu) );

		// add bias
		CUDNN_CHECK( hipdnnSetTensor4dDescriptor(biasTensorDesc,
				tensorFormat,
				dataType,
				1,
				top->C,
				1,
				1) );
		alpha = float(1);
		beta  = float(1);
		CUDNN_CHECK( hipdnnAddTensor(cudnnHandle, CUDNN_ADD_SAME_C,
				&alpha,
				biasTensorDesc,
				biasBlob->data_gpu,
				&beta,
				topTensorDesc,
				top->data_gpu) );

		// free buffer
		if (sizeInBytes!=0)
		{
			CUDA_CHECK( hipFree(workSpace) );
		}

	}

	void Backward(const Blob_t *top, Blob_t *bottom)
	{

		float alpha = (float)1.0f;
		float beta = (float)0.0f;
		CUDNN_CHECK(hipdnnConvolutionBackwardBias(cudnnHandle,
				&alpha,
				topTensorDesc,
				top->diff_gpu,
				&beta,
				biasTensorDesc,
				biasBlob->diff_gpu));

		CUDNN_CHECK(hipdnnConvolutionBackwardFilter(cudnnHandle,
				&alpha,
				bottomTensorDesc,
				bottom->data_gpu,
				topTensorDesc,
				top->diff_gpu,
				convDesc,
				&beta,
				filterDesc,
				filtersBlob->diff_gpu));

		CUDNN_CHECK(hipdnnConvolutionBackwardData(cudnnHandle,
				&alpha,
				filterDesc,
				filtersBlob->data_gpu,
				topTensorDesc,
				top->diff_gpu,
				convDesc,
				&beta,
				bottomTensorDesc,
				bottom->diff_gpu));

	}
};

class ActivationParameter_t
{
public:
	hipdnnActivationMode_t cudnn_activation_mode;
};

class ActivationLayer_t : public Layer_t
{
public:
	ActivationParameter_t *cudnn_activation_params;

	ActivationLayer_t(const ActivationParameter_t *cudnn_activation_params_) {
		cudnn_activation_params = const_cast<ActivationParameter_t *>(cudnn_activation_params_);
	}

	~ActivationLayer_t() {

	}

	void Setup(const Blob_t *bottom, Blob_t *top) {
		CUDNN_CHECK( hipdnnSetTensor4dDescriptor(bottomTensorDesc,
				tensorFormat,
				dataType,
				bottom->N,
				bottom->C,
				bottom->H,
				bottom->W) );

		top->N = bottom->N;
		top->C = bottom->C;
		top->H = bottom->H;
		top->W = bottom->W;

		CUDNN_CHECK( hipdnnSetTensor4dDescriptor(topTensorDesc,
				tensorFormat,
				dataType,
				top->N,
				top->C,
				top->H,
				top->W) );

		top->allocate_gpu_data();
		top->allocate_gpu_diff();
	}

	void Forward(const Blob_t *bottom, Blob_t *top) {
		float alpha = (float)1.0f;
		float beta = (float)0.0f;
		CUDNN_CHECK( hipdnnActivationForward(cudnnHandle,
				cudnn_activation_params->cudnn_activation_mode,
				&alpha,
				bottomTensorDesc,
				bottom->data_gpu,
				&beta,
				topTensorDesc,
				top->data_gpu) );
	}

	void Backward(const Blob_t *top, Blob_t *bottom) {
		float alpha = (float)1.0f;
		float beta = (float)0.0f;
		CUDNN_CHECK( hipdnnActivationBackward( cudnnHandle,
				cudnn_activation_params->cudnn_activation_mode,
				&alpha,
				topTensorDesc,
				top->data_gpu,
				topTensorDesc,
				top->diff_gpu,
				bottomTensorDesc,
				bottom->data_gpu,
				&beta,
				bottomTensorDesc,
				bottom->diff_gpu) );
	}
};

class PoolingParameter_t
{
public:
	hipdnnPoolingMode_t cudnn_pooling_mode;
	int poolsize_h;
	int poolsize_w;
	int pad_h;
	int pad_w;
	int stride_h;
	int stride_w;
};

class PoolingLayer_t : public Layer_t
{
public:
	PoolingParameter_t *cudnn_pooling_params;
	hipdnnPoolingDescriptor_t poolingDesc;

	PoolingLayer_t(const PoolingParameter_t *cudnn_pooling_params_) {
		cudnn_pooling_params = const_cast<PoolingParameter_t *>(cudnn_pooling_params_);

		CUDNN_CHECK( hipdnnCreatePoolingDescriptor(&poolingDesc) );
	}

	~PoolingLayer_t() {
		CUDNN_CHECK( hipdnnDestroyPoolingDescriptor(poolingDesc) );
	}

	void Setup(const Blob_t *bottom, Blob_t *top) {
		CUDNN_CHECK( hipdnnSetPooling2dDescriptor(poolingDesc,
				cudnn_pooling_params->cudnn_pooling_mode,
				cudnn_pooling_params->poolsize_h, // window
				cudnn_pooling_params->poolsize_w,
				cudnn_pooling_params->pad_h,
				cudnn_pooling_params->pad_w,
				cudnn_pooling_params->stride_h,
				cudnn_pooling_params->stride_w) );
		CUDNN_CHECK( hipdnnSetTensor4dDescriptor(bottomTensorDesc,
				tensorFormat,
				dataType,
				bottom->N,
				bottom->C,
				bottom->H,
				bottom->W) );
		/*
		CUDNN_CHECK( hipdnnGetPooling2dForwardOutputDim(poolingDesc,
				bottomTensorDesc,
				&(top->N),
				&(top->C),
				&(top->H),
				&(top->W)) );
		 */
		top->N = bottom->N;
		top->C = bottom->C;
		top->H = 1 + ceil((bottom->H + 2 * cudnn_pooling_params->pad_h - cudnn_pooling_params->poolsize_h) / cudnn_pooling_params->stride_h);
		top->W = 1 + ceil((bottom->W + 2 * cudnn_pooling_params->pad_w - cudnn_pooling_params->poolsize_w) / cudnn_pooling_params->stride_w);

		CUDNN_CHECK( hipdnnSetTensor4dDescriptor(topTensorDesc,
				tensorFormat,
				dataType,
				top->N,
				top->C,
				top->H,
				top->W) );

		top->allocate_gpu_data();
		top->allocate_gpu_diff();
	}

	void Forward(const Blob_t *bottom, Blob_t *top) {
		float alpha = (float)1.0f;
		float beta = (float)0.0f;
		CUDNN_CHECK( hipdnnPoolingForward(cudnnHandle,
				poolingDesc,
				&alpha,
				bottomTensorDesc,
				bottom->data_gpu,
				&beta,
				topTensorDesc,
				top->data_gpu) );
	}

	void Backward(const Blob_t *top, Blob_t *bottom) {
		float alpha = (float)1.0f;
		float beta = (float)0.0f;
		CUDNN_CHECK( hipdnnPoolingBackward( cudnnHandle,
				poolingDesc,
				&alpha,
				topTensorDesc,
				top->data_gpu,
				topTensorDesc,
				top->diff_gpu,
				bottomTensorDesc,
				bottom->data_gpu,
				&beta,
				bottomTensorDesc,
				bottom->diff_gpu) );
	}
};

class FullyConnectedParameter_t
{
public:
	int hidden_size;
};

class FullyConnectedLayer_t
{
public:
	hipblasHandle_t cublashandle;
	FullyConnectedParameter_t *fc_params;
	Blob_t *filtersBlob;
	Blob_t *biasBlob;
	Blob_t *bias_multiplier;
	int M_;
	int N_;
	int K_;
	FullyConnectedLayer_t(const FullyConnectedParameter_t *fc_params_) {
		fc_params = const_cast<FullyConnectedParameter_t *>(fc_params_);

		filtersBlob = NULL;
		biasBlob = NULL;
		bias_multiplier = NULL;

		M_ = 0;
		N_ = 0;
		K_ = 0;

		cublashandle = NULL;

		CUBLAS_CHECK( hipblasCreate(&cublashandle) );
	}

	~FullyConnectedLayer_t() {
		delete filtersBlob; filtersBlob = NULL;
		delete biasBlob; biasBlob = NULL;
		delete bias_multiplier; bias_multiplier = NULL;

		CUBLAS_CHECK( hipblasDestroy(cublashandle) );
	}

	void Setup(const Blob_t *bottom, Blob_t *top) {
		N_ = fc_params->hidden_size;
		K_ = bottom->C * bottom->H * bottom->W;
		M_ = bottom->N;
		filtersBlob = new Blob_t(1, 1, N_, K_);
		biasBlob = new Blob_t(1,1,1,N_);
		bias_multiplier = new Blob_t(1,1,1,M_);

		filtersBlob->allocate_gpu_data();
		filtersBlob->allocate_gpu_diff();
		biasBlob->allocate_gpu_data();
		biasBlob->allocate_gpu_diff();

		bias_multiplier->allocate_gpu_data();
		CUDA_CHECK( hipMemset(bias_multiplier->data_gpu, (float)1.0f, M_ * sizeof(float)) );
		//bias_multiplier->allocate_gpu_diff();

		top->N = bottom->N;
		top->C = N_;
		top->H = 1;
		top->W = 1;
		top->allocate_gpu_data();
		top->allocate_gpu_diff();

	}

	void Forward(const Blob_t *bottom, Blob_t *top) {
		gpu_gemm(cublashandle, CblasNoTrans, CblasTrans, M_, N_, K_, (float)1.,
				bottom->data_gpu, filtersBlob->data_gpu, (float)0., top->data_gpu);
		gpu_gemm(cublashandle, CblasNoTrans, CblasNoTrans, M_, N_, 1, (float)1.,
				bias_multiplier->data_gpu, biasBlob->data_gpu, (float)1., top->data_gpu);

	}

	void Backward(const Blob_t *top, Blob_t *bottom) {
		// Gradient with respect to weight
		gpu_gemm(cublashandle, CblasTrans, CblasNoTrans, N_, K_, M_, (float)1.,
				top->diff_gpu, bottom->data_gpu, (float)0., filtersBlob->diff_gpu);
		// Gradient with respect to bias
		gpu_gemv(cublashandle, CblasTrans, M_, N_, (float)1.,
				top->diff_gpu, bias_multiplier->data_gpu, (float)0., biasBlob->diff_gpu);
		// Gradient with respect to bottom data
		gpu_gemm(cublashandle, CblasNoTrans, CblasNoTrans, M_, K_, N_, (float)1.,
				top->diff_gpu, filtersBlob->data_gpu, (float)0., bottom->diff_gpu);
	}
};

class SoftmaxParameter_t
{
public:
	hipdnnSoftmaxAlgorithm_t cudnn_softmax_algo;
	hipdnnSoftmaxMode_t cudnn_softmax_mode;
};

class SoftmaxLayer_t : public Layer_t
{
public:
	SoftmaxParameter_t *cudnn_softmax_params;
	SoftmaxLayer_t(const SoftmaxParameter_t *cudnn_softmax_params_) {
		cudnn_softmax_params = const_cast<SoftmaxParameter_t *>(cudnn_softmax_params_);
	}

	~SoftmaxLayer_t() {

	}

	void Setup(const Blob_t *bottom, Blob_t *top) {
		CUDNN_CHECK( hipdnnSetTensor4dDescriptor(bottomTensorDesc,
				tensorFormat,
				dataType,
				bottom->N,
				bottom->C,
				bottom->H,
				bottom->W) );

		top->N = bottom->N;
		top->C = bottom->C;
		top->H = bottom->H;
		top->W = bottom->W;
		CUDNN_CHECK( hipdnnSetTensor4dDescriptor(topTensorDesc,
				tensorFormat,
				dataType,
				top->N,
				top->C,
				top->H,
				top->W) );

		top->allocate_gpu_data();
		top->allocate_gpu_diff();
	}

	void Forward(const Blob_t *bottom, Blob_t *top) {
		float alpha = (float)1.0f;
		float beta = (float)0.0f;
		CUDNN_CHECK( hipdnnSoftmaxForward(cudnnHandle,
				cudnn_softmax_params->cudnn_softmax_algo ,
				cudnn_softmax_params->cudnn_softmax_mode,
				&alpha,
				bottomTensorDesc,
				bottom->data_gpu,
				&beta,
				topTensorDesc,
				top->data_gpu) );
		top->data_to_cpu();
	}

	void Backward(const Blob_t *top, Blob_t *bottom) {
		float alpha = (float)1.0f;
		float beta = (float)0.0f;
		CUDNN_CHECK( hipdnnSoftmaxBackward( cudnnHandle,
				cudnn_softmax_params->cudnn_softmax_algo ,
				cudnn_softmax_params->cudnn_softmax_mode,
				&alpha,
				topTensorDesc,
				top->data_gpu,
				topTensorDesc,
				top->diff_gpu,
				&beta,
				bottomTensorDesc,
				bottom->diff_gpu) );
	}
};

class MultinomialLogisticLossParameter_t
{
public:
	std::vector<int> ignore_labels;
	bool normalize;
};

class MultinomialLogisticLossLayer_t
{
public:
	MultinomialLogisticLossParameter_t *mlr_params;


	MultinomialLogisticLossLayer_t(const MultinomialLogisticLossParameter_t *mlr_params_) {
		mlr_params = const_cast<MultinomialLogisticLossParameter_t *>(mlr_params_);
	}

	~MultinomialLogisticLossLayer_t() {

	}

	void Setup(const Blob_t *bottom, Blob_t *top) {
		top->N = 1;
		top->C = 1;
		top->H = 1;
		top->W = 1;
		top->allocate_cpu_data();
		top->allocate_cpu_diff();
		top->data_cpu[0] = 1.0f;
	}

	void Forward(const Blob_t *bottom, const Blob_t *label, Blob_t *top) {
		int num = bottom->N;
		int dim = bottom->count() / bottom->N;
		float loss = 0.0f;
		for (int i = 0; i < num; ++i) {
			int truelabel = static_cast<int>(label->data_cpu[i]);
			float prob = std::max(bottom->data_cpu[i * dim + truelabel], (float)1e-20);
			loss -= log(prob);
		}
		top->data_cpu[0] = (loss / num);
	}

	void Backward(const Blob_t *top, const Blob_t *label, Blob_t *bottom) {
		if(bottom->diff_cpu == NULL) {
			bottom->allocate_cpu_diff();
		}
		int num = bottom->N;
		int dim = bottom->count() / bottom->N;
		const float scale = - top->diff_cpu[0] / num;
		for (int i = 0; i < num; ++i) {
			int truelabel = static_cast<int>(label->data_cpu[i]);
			float prob = std::max(bottom->data_cpu[i * dim + truelabel], (float)1e-20);
			bottom->diff_cpu[i * dim + truelabel] = scale / prob;
		}
		bottom->diff_to_gpu();
	}
};

class Network_t
{
public:
	string net_name;
	int gpu_id;
	hipStream_t curand_stream;
	hiprandGenerator_t curand_generator;
	hiprandRngType_t curand_rngtype;
	hipblasHandle_t cublas_handle;

	Blob_t *batch_samples;
	Blob_t *batch_labels;

	ConvolutionParameter_t *conv1_params;
	ConvolutionLayer_t *conv1;
	Blob_t *conv1_top;

	ActivationParameter_t *relu1_params;
	ActivationLayer_t *relu1;
	Blob_t *relu1_top;

	PoolingParameter_t *mp1_params;
	PoolingLayer_t *mp1;
	Blob_t *mp1_top;

	FullyConnectedParameter_t *ip1_params;
	FullyConnectedLayer_t *ip1;
	Blob_t *ip1_top;

	SoftmaxParameter_t *sm1_params;
	SoftmaxLayer_t *sm1;
	Blob_t *sm1_top;

	MultinomialLogisticLossParameter_t *mlr1_params;
	MultinomialLogisticLossLayer_t *mlr1;
	Blob_t *mlr1_top;


	Blob_t *conv1_filtersBlob_old;
	Blob_t *conv1_biasBlob_old;
	Blob_t *ip1_filtersBlob_old;
	Blob_t *ip1_biasBlob_old;


	Network_t(string net_name_, int gpu_id_ = 0) {
		net_name = net_name_;
		gpu_id = gpu_id_;
		curand_stream = NULL;
		curand_generator = NULL;
		curand_rngtype = HIPRAND_RNG_PSEUDO_DEFAULT;
		cublas_handle = NULL;

		batch_samples = NULL;
		batch_labels = NULL;

		conv1 = NULL;
		conv1_top = NULL;
		conv1_params = NULL;
		relu1 = NULL;
		relu1_top = NULL;
		relu1_params = NULL;
		mp1 = NULL;
		mp1_top = NULL;
		mp1_params = NULL;
		ip1 = NULL;
		ip1_top = NULL;
		ip1_params = NULL;
		sm1 = NULL;
		sm1_top = NULL;
		sm1_params = NULL;
		mlr1 = NULL;
		mlr1_top = NULL;
		mlr1_params = NULL;


		conv1_filtersBlob_old = NULL;
		conv1_biasBlob_old = NULL;
		ip1_filtersBlob_old = NULL;
		ip1_biasBlob_old = NULL;

	}

	~Network_t() {
	}

	void DestroyNet() {

		hipSetDevice(gpu_id);

		delete batch_samples; batch_samples = NULL;
		delete batch_labels; batch_labels = NULL;

		delete conv1; conv1 = NULL;
		delete relu1; relu1 = NULL;
		delete mp1; mp1 = NULL;
		delete ip1; ip1 = NULL;
		delete sm1; sm1 = NULL;
		delete mlr1; mlr1 = NULL;

		delete conv1_top; conv1_top = NULL;
		delete relu1_top; relu1_top = NULL;
		delete mp1_top; mp1_top = NULL;
		delete ip1_top; ip1_top = NULL;
		delete sm1_top; sm1_top = NULL;
		delete mlr1_top; mlr1_top = NULL;

		delete conv1_params; conv1_params = NULL;
		delete relu1_params; relu1_params = NULL;
		delete mp1_params; mp1_params = NULL;
		delete ip1_params; ip1_params = NULL;
		delete sm1_params; sm1_params = NULL;
		delete mlr1_params; mlr1_params = NULL;

		delete conv1_filtersBlob_old; conv1_filtersBlob_old = NULL;
		delete conv1_biasBlob_old; conv1_biasBlob_old = NULL;
		delete ip1_filtersBlob_old; ip1_filtersBlob_old = NULL;
		delete ip1_biasBlob_old; ip1_biasBlob_old = NULL;

		CURAND_CHECK( hiprandDestroyGenerator(curand_generator) );
		CUDA_CHECK( hipStreamDestroy(curand_stream) );
		CUBLAS_CHECK( hipblasDestroy(cublas_handle) );
	}

	void BuildNet(int batch_size_, const string &net_params_file = "") {
		hipSetDevice(gpu_id);
		CUDA_CHECK( hipStreamCreate(&curand_stream) );
		curand_rngtype = HIPRAND_RNG_PSEUDO_DEFAULT;
		CURAND_CHECK( hiprandCreateGenerator(&curand_generator, curand_rngtype) );
		CURAND_CHECK( hiprandSetStream(curand_generator, curand_stream) );
		CUBLAS_CHECK( hipblasCreate(&cublas_handle) );

		batch_samples = new Blob_t(batch_size_, 3, 32, 32);
		batch_labels = new Blob_t(batch_size_, 1, 1, 1);
		batch_samples->allocate_gpu_data();
		batch_samples->allocate_gpu_diff();
		batch_labels->allocate_gpu_data();

		printf("conv1 setup.\n");
		conv1_top = new Blob_t();
		conv1_params = new ConvolutionParameter_t();
		conv1_params->filter_N = 3;
		conv1_params->filter_C = 32;
		conv1_params->filter_H = 5;
		conv1_params->filter_W = 5;
		conv1_params->pad_h = 0;
		conv1_params->pad_w = 0;
		conv1_params->stride_h = 1;
		conv1_params->stride_w = 1;
		conv1_params->upscale_h = 1;
		conv1_params->upscale_w = 1;
		conv1_params->cudnn_conv_mode = HIPDNN_CROSS_CORRELATION;
		conv1 = new ConvolutionLayer_t(conv1_params);
		CURAND_CHECK( hiprandGenerateNormal(curand_generator, conv1->filtersBlob->data_gpu, conv1->filtersBlob->count(), (float)0.0f, (float)0.01f) );
		CURAND_CHECK( hiprandGenerateNormal(curand_generator, conv1->biasBlob->data_gpu, conv1->biasBlob->count(), (float)0.0f, (float)0.01f) );
		conv1->Setup(batch_samples, conv1_top);


		printf("relu1 setup.\n");
		relu1_top = new Blob_t();
		relu1_params = new ActivationParameter_t();
		relu1_params->cudnn_activation_mode = HIPDNN_ACTIVATION_RELU;
		relu1 = new ActivationLayer_t(relu1_params);
		relu1->Setup(conv1_top, relu1_top);

		printf("mp1 setup.\n");
		mp1_top = new Blob_t();
		mp1_params = new PoolingParameter_t();
		mp1_params->cudnn_pooling_mode = HIPDNN_POOLING_MAX;
		mp1_params->poolsize_h = 2;
		mp1_params->poolsize_w = 2;
		mp1_params->pad_h = 0;
		mp1_params->pad_w = 0;
		mp1_params->stride_h = 2;
		mp1_params->stride_w = 2;
		mp1 = new PoolingLayer_t(mp1_params);
		mp1->Setup(relu1_top, mp1_top);

		printf("ip1 setup.\n");
		ip1_top = new Blob_t();
		ip1_params = new FullyConnectedParameter_t();
		ip1_params->hidden_size = 10;
		ip1 = new FullyConnectedLayer_t(ip1_params);
		ip1->Setup(mp1_top, ip1_top);
		CURAND_CHECK( hiprandGenerateNormal(curand_generator, ip1->filtersBlob->data_gpu, ip1->filtersBlob->count(), (float)0.0f, (float)0.01f) );
		CURAND_CHECK( hiprandGenerateNormal(curand_generator, ip1->biasBlob->data_gpu, ip1->biasBlob->count(), (float)0.0f, (float)0.01f) );


		printf("sm1 setup.\n");
		sm1_top = new Blob_t();
		sm1_params = new SoftmaxParameter_t();
		sm1_params->cudnn_softmax_algo = HIPDNN_SOFTMAX_ACCURATE;
		sm1_params->cudnn_softmax_mode = HIPDNN_SOFTMAX_MODE_CHANNEL;
		sm1 = new SoftmaxLayer_t(sm1_params);
		sm1->Setup(ip1_top, sm1_top);

		printf("mlr1 setup (in cpu).\n");
		mlr1_top = new Blob_t();
		mlr1_params = new MultinomialLogisticLossParameter_t();
		mlr1 = new MultinomialLogisticLossLayer_t(mlr1_params);
		mlr1->Setup(sm1_top, mlr1_top);

		printf("initialize old net params.\n");
		conv1_filtersBlob_old = new Blob_t(conv1->filtersBlob->N, conv1->filtersBlob->C, conv1->filtersBlob->H, conv1->filtersBlob->W);
		conv1_biasBlob_old = new Blob_t(conv1->biasBlob->N, conv1->biasBlob->C, conv1->biasBlob->H, conv1->biasBlob->W);
		conv1_filtersBlob_old->allocate_gpu_data();
		conv1_biasBlob_old->allocate_gpu_data();
		gpu_fill(NULL, conv1_filtersBlob_old->data_gpu, conv1_filtersBlob_old->count(), 0.0f, 0.0f);
		gpu_fill(NULL, conv1_biasBlob_old->data_gpu, conv1_biasBlob_old->count(), 0.0f, 0.0f);

		ip1_filtersBlob_old = new Blob_t(ip1->filtersBlob->N, ip1->filtersBlob->C, ip1->filtersBlob->H, ip1->filtersBlob->W);
		ip1_biasBlob_old = new Blob_t(ip1->biasBlob->N, ip1->biasBlob->C, ip1->biasBlob->H, ip1->biasBlob->W);
		ip1_filtersBlob_old->allocate_gpu_data();
		ip1_biasBlob_old->allocate_gpu_data();
		gpu_fill(NULL, ip1_filtersBlob_old->data_gpu, ip1_filtersBlob_old->count(), 0.0f, 0.0f);
		gpu_fill(NULL, ip1_biasBlob_old->data_gpu, ip1_biasBlob_old->count(), 0.0f, 0.0f);

		printf("build net (done).\n");
	}

	float Forward() {
		hipSetDevice(gpu_id);

		float loss = 0.0f;

		// printf("conv1 forward.\n");
		conv1->Forward(batch_samples, conv1_top);

		// printf("relu1 forward.\n");
		relu1->Forward(conv1_top, relu1_top);

		// printf("mp1 forward.\n");
		mp1->Forward(relu1_top, mp1_top);

		// printf("ip1 forward.\n");
		ip1->Forward(mp1_top, ip1_top);

		// printf("sm1 forward.\n");
		sm1->Forward(ip1_top, sm1_top);

		// printf("mlr1 forward.\n");
		mlr1->Forward(sm1_top, batch_labels, mlr1_top);

		loss = mlr1_top->data_cpu[0];

		return loss;
	}

	void Backward() {
		hipSetDevice(gpu_id);

		// printf("mlr1 backward.\n");
		mlr1->Backward(mlr1_top, batch_labels, sm1_top);

		// printf("sm1 backward.\n");
		sm1->Backward(sm1_top, ip1_top);

		// printf("ip1 backward.\n");
		ip1->Backward(ip1_top, mp1_top);

		// printf("mp1 backward.\n");
		mp1->Backward(mp1_top, relu1_top);

		// printf("relu1 backward.\n");
		relu1->Backward(relu1_top, conv1_top);

		// printf("conv1 backward.\n");
		conv1->Backward(conv1_top, batch_samples);
	}

	float ForwardBackward() {
		float loss = Forward();
		Backward();
		return loss;
	}

	void ComputeUpdateValueSingle(Blob_t *param_gradient_blob, Blob_t *param_blob_old,
			float lr_rate, float momentum, float weight_decay) {
		gpu_axpy(cublas_handle,
				param_gradient_blob->count(), weight_decay,
				param_gradient_blob->data_gpu,
				param_gradient_blob->diff_gpu);

		gpu_axpby(cublas_handle,
				param_gradient_blob->count(), lr_rate,
				param_gradient_blob->diff_gpu, momentum,
				param_blob_old->data_gpu);
		// copy
		gpu_copy(param_gradient_blob->count(),
				param_blob_old->data_gpu,
				param_gradient_blob->diff_gpu);
	}
	void ComputeUpdateValue(float lr_rate, float momentum, float weight_decay) {
		hipSetDevice(gpu_id);
		ComputeUpdateValueSingle(conv1->filtersBlob, conv1_filtersBlob_old, lr_rate, momentum, weight_decay);
		ComputeUpdateValueSingle(conv1->biasBlob, conv1_biasBlob_old, lr_rate, momentum, weight_decay);
		ComputeUpdateValueSingle(ip1->filtersBlob, ip1_filtersBlob_old, lr_rate, momentum, weight_decay);
		ComputeUpdateValueSingle(ip1->biasBlob, ip1_biasBlob_old, lr_rate, momentum, weight_decay);
	}

	void UpdateNet() {
		hipSetDevice(gpu_id);
		gpu_axpy(cublas_handle, conv1->filtersBlob->count(), float(-1), conv1->filtersBlob->diff_gpu, conv1->filtersBlob->data_gpu);
		gpu_axpy(cublas_handle, conv1->biasBlob->count(), float(-1), conv1->biasBlob->diff_gpu, conv1->biasBlob->data_gpu);
		gpu_axpy(cublas_handle, ip1->filtersBlob->count(), float(-1), ip1->filtersBlob->diff_gpu, ip1->filtersBlob->data_gpu);
		gpu_axpy(cublas_handle, ip1->biasBlob->count(), float(-1), ip1->biasBlob->diff_gpu, ip1->biasBlob->data_gpu);
	}

	void CopyNetParamsFrom(const Network_t *other) {
		CopyBlobData_gpu(other->conv1->filtersBlob, other->gpu_id, conv1->filtersBlob, gpu_id);
		CopyBlobData_gpu(other->conv1->biasBlob, other->gpu_id, conv1->biasBlob, gpu_id);
		CopyBlobData_gpu(other->ip1->filtersBlob, other->gpu_id, ip1->filtersBlob, gpu_id);
		CopyBlobData_gpu(other->ip1->biasBlob, other->gpu_id, ip1->biasBlob, gpu_id);
	}

	void AddNetParamsDiffFrom(const Network_t *other) {
		AddBlobDiff_gpu(other->conv1->filtersBlob, other->gpu_id, conv1->filtersBlob, gpu_id);
		AddBlobDiff_gpu(other->conv1->biasBlob, other->gpu_id, conv1->biasBlob, gpu_id);
		AddBlobDiff_gpu(other->ip1->filtersBlob, other->gpu_id, ip1->filtersBlob, gpu_id);
		AddBlobDiff_gpu(other->ip1->biasBlob, other->gpu_id, ip1->biasBlob, gpu_id);
	}

	void ClearNetParamsDiff() {
		hipSetDevice(gpu_id);
		CUDA_CHECK( hipMemset(conv1->filtersBlob->diff_gpu, 0, conv1->filtersBlob->count() * sizeof(float)) );
		CUDA_CHECK( hipMemset(conv1->biasBlob->diff_gpu, 0, conv1->biasBlob->count() * sizeof(float)) );
		CUDA_CHECK( hipMemset(ip1->filtersBlob->diff_gpu, 0, ip1->filtersBlob->count() * sizeof(float)) );
		CUDA_CHECK( hipMemset(ip1->biasBlob->diff_gpu, 0, ip1->biasBlob->count() * sizeof(float)) );
	}

};

struct thread_data_t
{
public:
	Blob_t *batch_samples;
	Blob_t *batch_labels;
	Network_t *net;
	int current_gpu_id;
	int net_gpu_id;
	float lr_rate;
	float momentum;
	float weight_decay;
};

void do_slave(void *data_)
{
	thread_data_t *data = (thread_data_t *)data_;
	hipSetDevice(data->net_gpu_id);
	CUDA_CHECK( hipMemcpy(data->net->batch_samples->data_gpu, data->batch_samples->data_cpu, data->batch_samples->count() * sizeof(float), hipMemcpyHostToDevice) );
	CUDA_CHECK( hipMemcpy(data->net->batch_labels->data_gpu, data->batch_labels->data_cpu, data->batch_labels->count() * sizeof(float), hipMemcpyHostToDevice) );
	float trn_loss = data->net->ForwardBackward();
	printf("trn_loss: %.6f\n", trn_loss);
	data->net->ComputeUpdateValue(data->lr_rate, data->momentum, data->weight_decay);
}


int main_test_data_layer_ok(int argc, char *argv[]) {
	if(argc != 12) {
		printf("Usage: <filename> trn_db_filename tst_db_filename mean_file lr_rate lr_stepsize momentum weight_decay trn_batch_size tst_batch_size max_epoch_num gpu_ids\n");
		return -1;
	}
	string trn_db_filename = string(argv[1]);
	string tst_db_filename = string(argv[2]);
	string mean_file = string(argv[3]);
	float lr_rate = atof(argv[4]);
	int lr_stepsize = atoi(argv[5]);
	float momentum = atof(argv[6]);
	float weight_decay = atof(argv[7]);
	int trn_batch_size = atoi(argv[8]);
	int tst_batch_size = atoi(argv[9]);
	int max_epoch_num = atoi(argv[10]);
	string gpu_ids_str = string(argv[11]);

	Blob_t *batch_samples = new Blob_t();
	Blob_t *batch_labels = new Blob_t();
	DataLayerParameter_t *data_param = new DataLayerParameter_t();
	data_param->backend = "lmdb";
	data_param->batch_size = trn_batch_size;
	data_param->source = trn_db_filename;
	data_param->mean_file = mean_file;
	DataLayer_t *trn_data_layer = new DataLayer_t(data_param);
	trn_data_layer->Setup();
	printf("forward datalayer.\n");
	trn_data_layer->Forward_cpu(batch_samples, batch_labels);
	printf("forward datalayer(done).\n");

	batch_samples->print_cpu_data(100);
	batch_labels->print_cpu_data(100);

	delete data_param; data_param = NULL;
	delete batch_samples; batch_samples = NULL;
	delete batch_labels; batch_labels = NULL;
	delete trn_data_layer; trn_data_layer = NULL;
	return 0;
}

int main_test_multigpu_ok(int argc, char *argv[]) {
	if(argc != 12) {
		printf("Usage: <filename> trn_db_filename tst_db_filename mean_file lr_rate lr_stepsize momentum weight_decay trn_batch_size tst_batch_size max_epoch_num gpu_ids\n");
		return -1;
	}
	string trn_db_filename = string(argv[1]);
	string tst_db_filename = string(argv[2]);
	string mean_file = string(argv[3]);
	float lr_rate = atof(argv[4]);
	int lr_stepsize = atoi(argv[5]);
	float momentum = atof(argv[6]);
	float weight_decay = atof(argv[7]);
	int trn_batch_size = atoi(argv[8]);
	int tst_batch_size = atoi(argv[9]);
	int max_epoch_num = atoi(argv[10]);
	string gpu_ids_str = string(argv[11]);


	int current_gpu_id;
	hipGetDevice(&current_gpu_id);
	printf("current gpu id: %d\n", current_gpu_id);

	vector<int> gpus;
	vector<string> strings;
	boost::split(strings, gpu_ids_str, boost::is_any_of(","));
	for (int i = 0; i < strings.size(); ++i) {
		gpus.push_back(boost::lexical_cast<int>(strings[i]));
	}
	int num_gpus = 0;
	hipGetDeviceCount(&num_gpus);
	printf("number of manually-set gpus: %ld, total %d gpus.\n", gpus.size(), num_gpus);

	if(num_gpus >= gpus.size()) {
		printf("enable P2P: ");
		EnableP2P(gpus);
		printf("%s \n", hipGetErrorString(hipGetLastError()));
	} else {
		gpus.clear();
		gpus.push_back(current_gpu_id);
	}

	hipSetDevice(current_gpu_id);

	vector<Network_t *> trn_nets(gpus.size());
	for(int i = 0; i < gpus.size(); i++) {
		trn_nets[i] = NULL;
	}
	printf("initialize nets for each gpu ...\n");
	for(int i = 0; i < gpus.size(); i++)
	{
		printf("=========== gpu [%d] ==============\n", gpus[i]);
		hipSetDevice(gpus[i]);
		trn_nets[i] = new Network_t(string("trn_nets_"+i), gpus[i]);
		trn_nets[i]->BuildNet(trn_batch_size, "");
		trn_nets[i]->batch_labels->allocate_cpu_data();
	}
	printf("initialize nets for each gpu (done) ...\n");

	hipSetDevice(current_gpu_id);

	pthread_t *threads;
	pthread_attr_t pta;
	threads = (pthread_t *) malloc(sizeof(pthread_t) * gpus.size());
	int ret_count = pthread_attr_init(&pta);
	thread_data_t thread_data[gpus.size()];

	// prepare batch data, should use blocking queue
	Blob_t *batch_samples = new Blob_t(trn_batch_size, 3, 32, 32);
	Blob_t *batch_labels  = new Blob_t(trn_batch_size, 1, 1, 1);
	batch_samples->allocate_cpu_data();
	batch_labels->allocate_cpu_data();
	for(int n = 0; n < batch_samples->N; n++) {
		for(int c = 0; c < batch_samples->C; c++) {
			for(int h = 0; h < batch_samples->H; h++) {
				for(int w = 0; w < batch_samples->W; w++) {
					int index = (((n)*batch_samples->C+c)*batch_samples->H+h)*batch_samples->W + w;
					batch_samples->data_cpu[index] = (float)rand() / (float)RAND_MAX;
				}
			}
		}
		batch_labels->data_cpu[n] = n;
	}

	for(int i = 0; i < gpus.size(); i++) {
		thread_data[i].lr_rate = lr_rate;
		thread_data[i].momentum = momentum;
		thread_data[i].weight_decay = weight_decay;
		thread_data[i].current_gpu_id = current_gpu_id;
		thread_data[i].net = trn_nets[i];
		thread_data[i].net_gpu_id = gpus[i];
		thread_data[i].batch_samples = batch_samples;
		thread_data[i].batch_labels = batch_labels;

		ret_count = pthread_create(&threads[i], &pta, (void*(*)(void*))do_slave, (void*)(&(thread_data[i])));
	}

	for(int i = 0; i < gpus.size(); i++) {
		ret_count = pthread_join(threads[i], NULL);
	}

	for(int i = 0; i < gpus.size(); i++) {
		hipSetDevice(gpus[i]);
		delete trn_nets[i]; trn_nets[i] = NULL;
	}

	hipSetDevice(current_gpu_id);
	delete batch_samples;
	delete batch_labels;

	if(num_gpus >= gpus.size()) {
		printf("disable P2P: ");
		DisableP2P(gpus);
		printf("%s \n", hipGetErrorString(hipGetLastError()));
	}
	hipDeviceReset();
	return 0;
}

int main(int argc, char *argv[]) {
	if(argc != 12) {
		printf("Usage: <filename> trn_db_filename tst_db_filename mean_file lr_rate lr_stepsize momentum weight_decay trn_batch_size tst_batch_size max_epoch_num gpu_ids\n");
		return -1;
	}
	string trn_db_filename = string(argv[1]);
	string tst_db_filename = string(argv[2]);
	string mean_file = string(argv[3]);
	float lr_rate = atof(argv[4]);
	int lr_stepsize = atoi(argv[5]);
	float momentum = atof(argv[6]);
	float weight_decay = atof(argv[7]);
	int trn_batch_size = atoi(argv[8]);
	int tst_batch_size = atoi(argv[9]);
	int max_epoch_num = atoi(argv[10]);
	string gpu_ids_str = string(argv[11]);


	int current_gpu_id;
	hipGetDevice(&current_gpu_id);
	printf("current gpu id: %d\n", current_gpu_id);

	vector<int> gpus;
	vector<string> strings;
	boost::split(strings, gpu_ids_str, boost::is_any_of(","));
	for (int i = 0; i < strings.size(); ++i) {
		gpus.push_back(boost::lexical_cast<int>(strings[i]));
	}
	int num_gpus = 0;
	hipGetDeviceCount(&num_gpus);
	printf("number of manually-set gpus: %ld, total %d gpus.\n", gpus.size(), num_gpus);

	if(num_gpus >= gpus.size()) {
		printf("enable P2P: ");
		EnableP2P(gpus);
		printf("%s \n", hipGetErrorString(hipGetLastError()));
	} else {
		gpus.clear();
		gpus.push_back(current_gpu_id);
	}

	hipSetDevice(current_gpu_id);

	vector<Network_t *> trn_nets(gpus.size());
	vector<Blob_t *> batch_samples_slices(gpus.size());
	vector<Blob_t *> batch_labels_slices(gpus.size());
	vector<int> batch_sizes(gpus.size());
	for(int i = 0; i < gpus.size(); i++) {
		trn_nets[i] = NULL;
		batch_samples_slices[i] = NULL;
		batch_labels_slices[i] = NULL;
		batch_sizes[i] = 0;
	}
	printf("initialize nets for each gpu ...\n");
	for(int i = 0; i < gpus.size(); i++)
	{
		printf("=========== gpu [%d] ==============\n", gpus[i]);
		hipSetDevice(gpus[i]);

		batch_samples_slices[i] = new Blob_t();
		batch_labels_slices[i] = new Blob_t();
		batch_sizes[i] = trn_batch_size / gpus.size();

		trn_nets[i] = new Network_t(string("trn_nets_"+i), gpus[i]);
		trn_nets[i]->BuildNet(batch_sizes[i], "");
		trn_nets[i]->batch_labels->allocate_cpu_data();
	}
	printf("initialize nets for each gpu (done) ...\n");

	hipSetDevice(current_gpu_id);

	Blob_t *trn_batch_samples = new Blob_t();
	Blob_t *trn_batch_labels = new Blob_t();
	DataLayerParameter_t *trn_data_param = new DataLayerParameter_t();
	trn_data_param->backend = "lmdb";
	trn_data_param->batch_size = trn_batch_size;
	trn_data_param->source = trn_db_filename;
	trn_data_param->mean_file = mean_file;
	DataLayer_t *trn_data_layer = new DataLayer_t(trn_data_param);
	trn_data_layer->Setup();

	Blob_t *tst_batch_samples = new Blob_t();
	Blob_t *tst_batch_labels = new Blob_t();
	DataLayerParameter_t *tst_data_param = new DataLayerParameter_t();
	tst_data_param->backend = "lmdb";
	tst_data_param->batch_size = tst_batch_size;
	tst_data_param->source = tst_db_filename;
	tst_data_param->mean_file = mean_file;
	DataLayer_t *tst_data_layer = new DataLayer_t(tst_data_param);
	tst_data_layer->Setup();

	Network_t *trn_net = new Network_t("trn_net", current_gpu_id);
	trn_net->BuildNet(trn_batch_size, "");
	trn_net->batch_labels->allocate_cpu_data();
	Network_t *tst_net = new Network_t("tst_net", current_gpu_id);
	tst_net->BuildNet(tst_batch_size, "");
	tst_net->batch_labels->allocate_cpu_data();

	pthread_t *threads;
	pthread_attr_t pta;
	threads = (pthread_t *) malloc(sizeof(pthread_t) * gpus.size());
	int ret_count = pthread_attr_init(&pta);
	thread_data_t thread_data[gpus.size()];
	for(int i = 0; i < gpus.size(); i++) {
		thread_data[i].lr_rate = lr_rate;
		thread_data[i].momentum = momentum;
		thread_data[i].weight_decay = weight_decay;
		thread_data[i].current_gpu_id = current_gpu_id;
		thread_data[i].net = trn_nets[i];
		thread_data[i].net_gpu_id = gpus[i];
		thread_data[i].batch_samples = batch_samples_slices[i];
		thread_data[i].batch_labels = batch_labels_slices[i];
	}

	for(int epoch = 0; epoch < max_epoch_num; epoch++) {

		// testing net
		float tst_loss = 0.0f;
		tst_net->CopyNetParamsFrom(trn_net);
		for(int iter = 0; iter < floor(10000 / tst_batch_size); iter++) {
			tst_data_layer->Forward_cpu(tst_batch_samples, tst_batch_labels);
			tst_loss += tst_net->Forward();
		}

		// training net
		for(int iter = 0; iter < floor(50000 / trn_batch_size); iter++) {
			trn_data_layer->Forward_cpu_multi(batch_samples_slices, batch_labels_slices, batch_sizes);

			trn_net->ClearNetParamsDiff();

			// copy trn_net params into trn_nets_i
			for(int i = 0; i < gpus.size(); i++) {
				trn_nets[i]->CopyNetParamsFrom(trn_net);
			}

			for(int i = 0; i < gpus.size(); i++) {
				ret_count = pthread_create(&threads[i], &pta, (void*(*)(void*))do_slave, (void*)(&(thread_data[i])));
			}

			for(int i = 0; i < gpus.size(); i++) {
				ret_count = pthread_join(threads[i], NULL);
			}

			hipDeviceSynchronize();
			hipSetDevice(current_gpu_id);
			// copy update values from each sub nets to the main trn_net
			for(int i = 0; i < gpus.size(); i++) {
				trn_net->AddNetParamsDiffFrom(trn_nets[i]);
			}
			trn_net->UpdateNet();
		}
	}

	for(int i = 0; i < gpus.size(); i++) {
		hipSetDevice(gpus[i]);
		delete trn_nets[i]; trn_nets[i] = NULL;
	}

	hipSetDevice(current_gpu_id);
	delete trn_batch_samples;
	delete trn_batch_labels;
	delete tst_batch_samples;
	delete tst_batch_labels;
	delete trn_net;
	delete tst_net;

	delete trn_data_param; trn_data_param = NULL;
	delete trn_data_layer; trn_data_layer = NULL;
	delete tst_data_param; tst_data_param = NULL;
	delete tst_data_layer; tst_data_layer = NULL;

	if(num_gpus >= gpus.size()) {
		printf("disable P2P: ");
		DisableP2P(gpus);
		printf("%s \n", hipGetErrorString(hipGetLastError()));
	}
	free(threads); threads = NULL;
	hipDeviceReset();
	exit(EXIT_SUCCESS);
}
