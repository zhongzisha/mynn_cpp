#include "hip/hip_runtime.h"



#include "common.hpp"
#include "blob.hpp"
#include "common_layer.hpp"
#include "data_layer.hpp"
#include "conv_layer.hpp"
#include "loss_layer.hpp"

#include "network_cifar10_convg.hpp"




int main(int argc, char **argv) {
	if(argc != 11) {
		printf("Usage: <filename> trn_db_filename tst_db_filename mean_file lr_rate lr_stepsize momentum weight_decay batch_size max_epoch_num gpu_ids\n");
		return -1;
	}
	string trn_db_filename = string(argv[1]);
	string tst_db_filename = string(argv[2]);
	string mean_file = string(argv[3]);
	float lr_rate = atof(argv[4]);
	int lr_stepsize = atoi(argv[5]);
	float momentum = atof(argv[6]);
	float weight_decay = atof(argv[7]);
	int batch_size = atoi(argv[8]);
	int max_epoch_num = atoi(argv[9]);
	string gpu_ids_str = string(argv[10]);

	int main_gpu_id = 0;
	hipSetDevice(main_gpu_id);
	DataLayerParameter_t *trn_data_param = new DataLayerParameter_t();
	trn_data_param->backend = "lmdb";
	trn_data_param->batch_size = batch_size;
	trn_data_param->source = trn_db_filename;
	trn_data_param->mean_file = mean_file;
	trn_data_param->crop_size = 0;
	trn_data_param->scale = 1.0f;
	trn_data_param->mirror = true;
	trn_data_param->has_mean_file = true;
	trn_data_param->phase = "train";
	DataLayer_t *trn_data_layer = new DataLayer_t(trn_data_param);
	trn_data_layer->Setup();

	DataLayerParameter_t *tst_data_param = new DataLayerParameter_t();
	tst_data_param->backend = "lmdb";
	tst_data_param->batch_size = batch_size;
	tst_data_param->source = tst_db_filename;
	tst_data_param->mean_file = mean_file;
	tst_data_param->crop_size = 0;
	tst_data_param->scale = 1.0f;
	tst_data_param->mirror = false;
	tst_data_param->has_mean_file = true;
	tst_data_param->phase = "test";
	DataLayer_t *tst_data_layer = new DataLayer_t(tst_data_param);
	tst_data_layer->Setup();

	Cifar10Network_convg_t *trn_net = new Cifar10Network_convg_t("trn_net", main_gpu_id);
	trn_net->BuildNet(batch_size, true, "");

	int num_tst_iters = ceil(10000 / batch_size);
	int num_trn_iters = ceil(50000 / batch_size);
	for(int epoch = 0; epoch < max_epoch_num; epoch++) {

		// testing net
		float tst_loss = 0.0f, tst_loss_batch = 0.0f;
		float tst_acc  = 0.0f, tst_acc_batch  = 0.0f;
		for(int iter = 0; iter < num_tst_iters; iter++) {
			tst_data_layer->Forward_to_Network(trn_net->batch_samples, trn_net->batch_labels);
			trn_net->Forward(&tst_loss_batch, &tst_acc_batch);
			tst_loss += tst_loss_batch;
			tst_acc += tst_acc_batch;
		}
		tst_loss /= num_tst_iters;
		tst_acc  /= num_tst_iters;

		// training net
		float trn_loss = 0.0f, trn_loss_batch = 0.0f;
		float trn_acc  = 0.0f, trn_acc_batch  = 0.0f;
		for(int iter = 0; iter < num_trn_iters; iter++) {
			trn_data_layer->Forward_to_Network(trn_net->batch_samples, trn_net->batch_labels);
			trn_net->ForwardBackward(&trn_loss_batch, &trn_acc_batch);
			trn_loss += trn_loss_batch;
			trn_acc  += trn_acc_batch;
			trn_net->ComputeUpdateValue(lr_rate, momentum, weight_decay);
			trn_net->UpdateNet();
		}
		trn_loss /= num_trn_iters;
		trn_acc  /= num_trn_iters;

		// update learning rate
		if((epoch != 0) && (epoch % lr_stepsize == 0))
		{
			lr_rate /= 10;
			trn_net->SaveNetParams(epoch);
		}
		printf("epoch[%d]: trn_loss=%.6f, trn_acc=%.6f, tst_loss=%.6f, tst_acc=%.6f\n",
				epoch, trn_loss, trn_acc, tst_loss, tst_acc);
	}

	delete trn_net;

	delete trn_data_layer;
	delete tst_data_layer;
	delete trn_data_param;
	delete tst_data_param;

	hipDeviceReset();
	return 0;
}


















