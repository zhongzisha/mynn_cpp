#include "hip/hip_runtime.h"

#include "common.hpp"

const char* curandGetErrorString(hiprandStatus_t error) {
	switch (error) {
	case HIPRAND_STATUS_SUCCESS:
		return "HIPRAND_STATUS_SUCCESS";
	case HIPRAND_STATUS_VERSION_MISMATCH:
		return "HIPRAND_STATUS_VERSION_MISMATCH";
	case HIPRAND_STATUS_NOT_INITIALIZED:
		return "HIPRAND_STATUS_NOT_INITIALIZED";
	case HIPRAND_STATUS_ALLOCATION_FAILED:
		return "HIPRAND_STATUS_ALLOCATION_FAILED";
	case HIPRAND_STATUS_TYPE_ERROR:
		return "HIPRAND_STATUS_TYPE_ERROR";
	case HIPRAND_STATUS_OUT_OF_RANGE:
		return "HIPRAND_STATUS_OUT_OF_RANGE";
	case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
		return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
	case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
		return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
	case HIPRAND_STATUS_LAUNCH_FAILURE:
		return "HIPRAND_STATUS_LAUNCH_FAILURE";
	case HIPRAND_STATUS_PREEXISTING_FAILURE:
		return "HIPRAND_STATUS_PREEXISTING_FAILURE";
	case HIPRAND_STATUS_INITIALIZATION_FAILED:
		return "HIPRAND_STATUS_INITIALIZATION_FAILED";
	case HIPRAND_STATUS_ARCH_MISMATCH:
		return "HIPRAND_STATUS_ARCH_MISMATCH";
	case HIPRAND_STATUS_INTERNAL_ERROR:
		return "HIPRAND_STATUS_INTERNAL_ERROR";
	}
	return "Unknown hiprand status";
}

const char* cublasGetErrorString(hipblasStatus_t error) {
	switch (error) {
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";
	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";
	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";
	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";
	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";
	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";
	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";
	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";
#if CUDA_VERSION >= 6000
	case HIPBLAS_STATUS_NOT_SUPPORTED:
		return "HIPBLAS_STATUS_NOT_SUPPORTED";
#endif
#if CUDA_VERSION >= 6050
	case HIPBLAS_STATUS_UNKNOWN:
		return "HIPBLAS_STATUS_UNKNOWN";
#endif
	}
	return "Unknown cublas status";
}

void EnableP2P(vector<int> gpus)
{
	// check p2p access
	hipDeviceProp_t prop[gpus.size()];
	for(int i = 0; i < gpus.size(); i++) {
		hipGetDeviceProperties(&prop[i], gpus[i]);
	}

	for(int i = 0; i < gpus.size(); i++) {
		for(int j = 0; j < gpus.size(); j++) {
			if(i==j)
				continue;
			int can_access_peer;
			hipDeviceCanAccessPeer(&can_access_peer, gpus[i], gpus[j]);
			if(can_access_peer) {
				hipSetDevice(gpus[i]);
				hipDeviceEnablePeerAccess(gpus[j], 0);
				hipSetDevice(gpus[j]);
				hipDeviceEnablePeerAccess(gpus[i], 0);
				const bool has_uva = (prop[gpus[i]].unifiedAddressing && prop[gpus[j]].unifiedAddressing);
				if(has_uva) {
					// printf("(%d <--> %d): YES!\n", gpus[i], gpus[j]);
					LOG(INFO) << gpus[i] << " <--> " << gpus[j] << ": UVA YES!\n";
				}
			} else {
				// printf("(%d <--> %d): NO!\n", gpus[i], gpus[j]);
				LOG(INFO) << gpus[i] << " <--> " << gpus[j] << ": UVA NO!\n";
			}
		}
	}
}

void DisableP2P(vector<int> gpus)
{
	for(int i = 0; i < gpus.size(); i++) {
		hipSetDevice(gpus[i]);
		hipDeviceDisablePeerAccess(gpus[i]);
	}
}

void cpu_add(const int N, const float *a, const float *b, float *y) {
#pragma omp parallel
	for(int i = 0; i < N; i++) {
		y[i] = a[i] + b[i];
	}
}

__global__ void add_kernel(const int n, const float* a,
		const float* b, float* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = a[index] + b[index];
	}
}

void gpu_add(const int N, const float* a, const float* b, float* y) {
	// NOLINT_NEXT_LINE(whitespace/operators)
	add_kernel<<<GPU_GET_BLOCKS(N), GPU_CUDA_NUM_THREADS>>>(N, a, b, y);
}

__global__ void set_kernel(const int n, const float alpha, float* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = alpha;
	}
}
void cpu_set(const int N, const float alpha, float *Y) {
#pragma omp parallel
	for(int i = 0; i < N; i++) {
		Y[i] = alpha;
	}
}
void gpu_set(const int N, const float alpha, float* Y) {
	set_kernel<<<GPU_GET_BLOCKS(N), GPU_CUDA_NUM_THREADS>>>(N, alpha, Y);
}

void gpu_copy(const int N, const float *X, float *Y) {
	CUDA_CHECK( hipMemcpy(Y, X, sizeof(float) * N, hipMemcpyDefault) );
}

void gpu_asum(hipblasHandle_t cublashandle, const int n, const float* x, float* y) {
	CUBLAS_CHECK(hipblasSasum(cublashandle, n, x, 1, y));
}

void gpu_scal(hipblasHandle_t cublashandle, const int N, const float alpha, float *X) {
	CUBLAS_CHECK( hipblasSscal(cublashandle, N, &alpha, X, 1) );
}

void gpu_axpy(hipblasHandle_t cublashandle, const int N, const float alpha,
		const float* X, float* Y) {
	CUBLAS_CHECK( hipblasSaxpy(cublashandle, N, &alpha, X, 1, Y, 1) );
}

void gpu_axpby(hipblasHandle_t cublashandle, const int N, const float alpha,
		const float* X, const float beta, float* Y) {
	gpu_scal(cublashandle, N, beta, Y);
	gpu_axpy(cublashandle, N, alpha, X, Y);
}

void gpu_gemv(hipblasHandle_t cublashandle,
		const CBLAS_TRANSPOSE TransA, const int M,
		const int N, const float alpha, const float* A, const float* x,
		const float beta, float* y) {
	hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	CUBLAS_CHECK( hipblasSgemv(cublashandle, cuTransA, N, M, &alpha, A, N, x, 1, &beta, y, 1) );
}

void gpu_gemm(hipblasHandle_t cublashandle,
		const CBLAS_TRANSPOSE TransA,
		const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
		const float alpha, const float* A, const float* B, const float beta,
		float* C) {
	// Note that cublas follows fortran order.
	int lda = (TransA == CblasNoTrans) ? K : M;
	int ldb = (TransB == CblasNoTrans) ? N : K;
	hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
	hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
	CUBLAS_CHECK( hipblasSgemm(cublashandle, cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N) );
}

void cpu_fill(float *ptr, int count, float value) {
	memset(ptr, value, count * sizeof(float));
}

void gpu_fill(hiprandGenerator_t curand_generator, float *ptr, int count, float mu, float std) {
	if(std == 0.0f) {
		CUDA_CHECK( hipMemset(ptr, mu, count * sizeof(float)) );
	} else {
		CURAND_CHECK( hiprandGenerateNormal(curand_generator, ptr, count, mu, std) );
	}
}
