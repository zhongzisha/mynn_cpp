#include "hip/hip_runtime.h"


#include "common.hpp"
#include "blob.hpp"
#include "common_layer.hpp"
#include "data_layer.hpp"
#include "conv_layer.hpp"
#include "loss_layer.hpp"
#include "network_cifar10.hpp"

int main(int argc, char **argv) {
	if(argc != 13) {
		printf("Usage: <filename> main_gpu_id db_backend trn_db_filename tst_db_filename mean_file lr_rate lr_stepsize momentum weight_decay batch_size max_epoch_num gpu_ids\n");
		return -1;
	}
	int main_gpu_id = atoi(argv[1]);
	string db_backend = string(argv[2]);
	string trn_db_filename = string(argv[3]);
	string tst_db_filename = string(argv[4]);
	string mean_file = string(argv[5]);
	float lr_rate = atof(argv[6]);
	int lr_stepsize = atoi(argv[7]);
	float momentum = atof(argv[8]);
	float weight_decay = atof(argv[9]);
	int batch_size = atoi(argv[10]);
	int max_epoch_num = atoi(argv[11]);
	string gpu_ids_str = string(argv[12]);


	hipSetDevice(main_gpu_id);
	DataLayerParameter_t *trn_data_param = new DataLayerParameter_t();
	trn_data_param->backend = db_backend;
	trn_data_param->batch_size = batch_size;
	trn_data_param->source = trn_db_filename;
	trn_data_param->mean_file = mean_file;
	trn_data_param->crop_size = 0;
	trn_data_param->scale = 1.0f;
	trn_data_param->mirror = true;
	trn_data_param->has_mean_file = true;
	trn_data_param->phase = "train";
	DataLayer_t *trn_data_layer = new DataLayer_t(trn_data_param);
	trn_data_layer->Setup();

	DataLayerParameter_t *tst_data_param = new DataLayerParameter_t();
	tst_data_param->backend = db_backend;
	tst_data_param->batch_size = batch_size;
	tst_data_param->source = tst_db_filename;
	tst_data_param->mean_file = mean_file;
	tst_data_param->crop_size = 0;
	tst_data_param->scale = 1.0f;
	tst_data_param->mirror = false;
	tst_data_param->has_mean_file = true;
	tst_data_param->phase = "test";
	DataLayer_t *tst_data_layer = new DataLayer_t(tst_data_param);
	tst_data_layer->Setup();

	Cifar10Network_t *trn_net = new Cifar10Network_t("trn_net", main_gpu_id);
	trn_net->BuildNet(batch_size, true, "");

	int num_tst_iters = ceil(10000 / batch_size);
	int num_trn_iters = ceil(50000 / batch_size);
	for(int epoch = 0; epoch < max_epoch_num; epoch++) {

		// testing net
		float tst_loss = 0.0f, tst_loss_batch = 0.0f;
		float tst_acc  = 0.0f, tst_acc_batch  = 0.0f;
		for(int iter = 0; iter < num_tst_iters; iter++) {
			tst_data_layer->Forward_to_Network(trn_net->batch_samples, trn_net->batch_labels);
			trn_net->Forward(&tst_loss_batch, &tst_acc_batch);
			tst_loss += tst_loss_batch;
			tst_acc += tst_acc_batch;
		}
		tst_loss /= num_tst_iters;
		tst_acc  /= num_tst_iters;
		LOG(INFO) << "epoch[" << epoch << "]: tst_loss=" << tst_loss << ", tst_acc=" << tst_acc << "\n";

		// training net
		float trn_loss = 0.0f, trn_loss_batch = 0.0f;
		float trn_acc  = 0.0f, trn_acc_batch  = 0.0f;
		for(int iter = 0; iter < num_trn_iters; iter++) {
			trn_data_layer->Forward_to_Network(trn_net->batch_samples, trn_net->batch_labels);
			trn_net->ForwardBackward(&trn_loss_batch, &trn_acc_batch);
			trn_loss += trn_loss_batch;
			trn_acc  += trn_acc_batch;
			trn_net->ComputeUpdateValue(lr_rate, momentum, weight_decay);
			trn_net->UpdateNet();
		}
		trn_loss /= num_trn_iters;
		trn_acc  /= num_trn_iters;

		// update learning rate
		if((epoch != 0) && (epoch % lr_stepsize == 0)) {
			lr_rate /= 10;
			// trn_net->SaveNetParams(epoch);
		}
		LOG(INFO) << "epoch[" << epoch << "]: trn_loss=" << trn_loss << ", trn_acc=" << trn_acc << "\n";
	}

	delete trn_net;

	delete trn_data_layer;
	delete tst_data_layer;
	delete trn_data_param;
	delete tst_data_param;

	hipDeviceReset();
	return 0;
}




















